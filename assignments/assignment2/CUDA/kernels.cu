#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define maxBlockDim 1024

__global__ void gpu_Heat (double *h, double *g, int N) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	// kernel computation
	if (i>0 && i< N - 1 && j>0 && j< N - 1)
	{
		g[i*N+j]= 0.25 * (h[ i*N + (j-1) ]+  // left
					     	h[ i*N + (j+1) ]+  // right
				         h[ (i-1)*N + j ]+  // top
				            h[ (i+1)*N + j ]); // bottom
	}
}

__global__ void gpu_Residual (double *h, double *g, int N, double *diff)
{	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
  int index = (N-2) * (i-1) + j-1;
	// kernel computation
	if (i>0 && i< N - 1 && j>0 && j< N - 1)
	{
		g[i*N+j]= 0.25 * (h[ i*N + (j-1) ]+  // left
					     	h[ i*N + (j+1) ]+  // right
				         h[ (i-1)*N + j ]+  // top
				            h[ (i+1)*N + j ]); // bottom
	
	  diff[index] = g[i*N + j] - h[i*N + j];
	  diff[index] = diff[index] * diff[index];
	}
}

__global__ void gpu_Reduction(double *g_idata, double *g_odata, int N) {
  __shared__ double sdata[maxBlockDim];
  unsigned int s;

  // Cada thread realiza la suma parcial de los datos que le
  // corresponden y la deja en la memoria compartida
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int gridSize = blockDim.x*2*gridDim.x;
  sdata[tid] = 0;
  while (i < N) {
    sdata[tid] += g_idata[i] + g_idata[i+blockDim.x];
    i += gridSize;
  }
  __syncthreads();

  // Hacemos la reduccion en la memoria compartida
  for (s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  // desenrrollamos el ultimo warp activo
  if (tid < 32) {
    volatile double *smem = sdata;

    smem[tid] += smem[tid + 32];
    smem[tid] += smem[tid + 16];
    smem[tid] += smem[tid + 8];
    smem[tid] += smem[tid + 4];
    smem[tid] += smem[tid + 2];
    smem[tid] += smem[tid + 1];
  }


  // El thread 0 escribe el resultado de este bloque en la memoria global
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

